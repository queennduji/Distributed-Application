//given by professor

/*
 To compile: 
    nvcc -arch=sm_60 reduction_kernel.cu
 To run with the array size 2^20, expo dist mean 5, and init seed 17:
    ./a.out 20 5 17
 */
#include <stdio.h>
#include <stdlib.h>

template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

template<class T>
T reduceCPU(T *data, int size)
{
    T sum = data[0];
    T c = (T)0.0;

    for (int i = 1; i < size; i++)
    {
        T y = data[i] - c;
        T t = sum + y;
        c = (t - sum) - y;
        sum = t;
    }

    return sum;
}

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

/*
    This version uses sequential addressing -- no divergence or bank conflicts.
*/
__global__ void
reduce(double *g_idata, double *g_odata, unsigned int n)
{
    double *sdata = SharedMemory<double>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

    sdata[tid] = (i < n) ? g_idata[i] : 0;

    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void mapqroot(double *inputA)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
   
    inputA[tid] = sqrt(sqrt(inputA[tid]));
}


// CUDA Runtime
#include <hip/hip_runtime.h>

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

extern "C" double reduction(int n, int mean, int seed)
{
    int size = 1<<n;    // number of elements to reduce//
    int maxThreads = 256;  // number of threads per block

    // create random input data on CPU
    unsigned int bytes = size * sizeof(double);

    double *h_idata = (double *) malloc(bytes);

    srand48(seed);
    for (int i=0; i<size; i++)
    {
            // h_idata[i] = 1.0; // for testing
            // expo dist with mean 5.0
            h_idata[i] = -mean * log(drand48());
    }

    int numBlocks = size / maxThreads;
    int numThreads = size;

    int smemSize = maxThreads * sizeof(double);

    // allocate mem for the result on host side
    double *h_odata = (double *) malloc(numBlocks*sizeof(double));

    // allocate device memory and data
    double  *d_idata = NULL;
    double *d_odata = NULL;

    checkCudaErrors(hipMalloc((void **) &d_idata, bytes));
    checkCudaErrors(hipMalloc((void **) &d_odata, numBlocks*sizeof(double)));

    // copy data directly to device memory
    checkCudaErrors(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));

    mapqroot<<<numBlocks,maxThreads>>>(d_idata);

    reduce<<<numBlocks,maxThreads,smemSize>>>(d_idata, d_odata, numThreads);

    int s=numBlocks;

    while (s > 1) {
        reduce<<<(s+maxThreads-1)/maxThreads,maxThreads,smemSize>>>(d_odata, d_odata, s);
        s = (s+maxThreads-1)/maxThreads;
    }

    checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(double), hipMemcpyDeviceToHost));        
    return h_odata[0];
}
